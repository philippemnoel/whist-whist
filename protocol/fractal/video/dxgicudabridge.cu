#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <libavcodec/avcodec.h>

#include "dxgicudabridge.h"

__global__ bool dxgi_cuda_transfer_data(CaptureDevice* device,
                                        video_encoder_t* encoder) {
    int i = 3;
    for (int j = 0; j < 52; ++j) {
        i += j;
    }
    return i;
}