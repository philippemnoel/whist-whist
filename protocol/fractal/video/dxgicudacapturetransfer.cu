#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#define WIN32_LEAN_AND_MEAN
#include <cuda_d3d11_interop.h>
#include <stdio.h>

#include "dxgicudacapturetransfer.h"

extern "C" {
bool cuda_is_available = false;
bool active_transfer_context = false;
hipGraphicsResource_t resource = NULL;

int dxgi_cuda_start_transfer_context(CaptureDevice* device) {
    static bool tried_cuda_check = false;
    if (!tried_cuda_check) {
        tried_cuda_check = true;

        if (hipDeviceSynchronize() != hipSuccess) {
            LOG_INFO("CUDA requested but not available on this device.");
            return 1;  // cuda unavailable
        } else {
            cuda_is_available = true;
        }
    }

    if (cuda_is_available) {
        if (active_transfer_context) return 0;

        hipError_t res = cudaGraphicsD3D11RegisterResource(
            &resource, device->screenshot.staging_texture, 0);
        if (res != hipSuccess) {
            LOG_ERROR("Error: %s | %s", hipGetErrorName(res),
                      hipGetErrorString(res));
            return -1;
        }

        active_transfer_context = true;

        return 0;
    } else {
        return 1;  // cuda unavailable
    }
}

void dxgi_cuda_close_transfer_context() {
    if (cuda_is_available && active_transfer_context && resource) {
        hipGraphicsUnregisterResource(resource);
        active_transfer_context = false;
    }
}

int dxgi_cuda_transfer_capture(CaptureDevice* device,
                               video_encoder_t* encoder) {
    if (cuda_is_available && active_transfer_context) {
        hipError_t res;
        hipArray_t mappedArray;

        static int times_measured = 0;
        static double time_spent = 0.;

        clock dxgi_cuda_transfer_timer;
        start_timer(&dxgi_cuda_transfer_timer);

        res = cudaGraphicsResourceSetMapFlags(resource,
                                              cudaGraphicsMapFlagsReadOnly);
        if (res != hipSuccess) {
            LOG_ERROR("Error: %s | %s", hipGetErrorName(res),
                      hipGetErrorString(res));
            return -1;
        }

        res = hipGraphicsMapResources(1, &resource, 0);
        if (res != hipSuccess) {
            LOG_ERROR("Error: %s | %s", hipGetErrorName(res),
                      hipGetErrorString(res));
            return -1;
        }

        res =
            hipGraphicsSubResourceGetMappedArray(&mappedArray, resource, 0, 0);
        if (res != hipSuccess) {
            LOG_ERROR("Error: %s | %s", hipGetErrorName(res),
                      hipGetErrorString(res));
            return -1;
        }

        res = hipGraphicsUnmapResources(1, &resource, 0);
        if (res != hipSuccess) {
            LOG_ERROR("Error: %s | %s", hipGetErrorName(res),
                      hipGetErrorString(res));
            return -1;
        }

        encoder->sw_frame->pts++;
        encoder->hw_frame->pict_type = encoder->sw_frame->pict_type;

        res = hipMemcpy2DFromArray(
            encoder->hw_frame->data[0], encoder->hw_frame->linesize[0],
            mappedArray, 0, 0, encoder->hw_frame->width * 4,
            encoder->hw_frame->height, hipMemcpyDefault);
        if (res != hipSuccess) {
            LOG_ERROR("Error: %s | %s", hipGetErrorName(res),
                      hipGetErrorString(res));
            return -1;
        }

        times_measured++;
        time_spent += get_timer(dxgi_cuda_transfer_timer);

        if (times_measured == 10) {
            LOG_INFO(
                "Average time transferring dxgi data to encoder frame on CUDA "
                "GPU: "
                "%f",
                time_spent / times_measured);
            times_measured = 0;
            time_spent = 0.0;
        }
        return 0;
    } else {
        return 1;  // cuda unavailable
    }
}
}