#include <hip/hip_runtime.h>
#define WIN32_LEAN_AND_MEAN
#include <cuda_d3d11_interop.h>
#include <stdio.h>

#include "dxgicudacapturetransfer.h"

extern "C" {
bool cuda_is_available = true;
hipGraphicsResource_t resource = NULL;

int dxgi_cuda_start_transfer_context(CaptureDevice* device) {
    static bool tried_cuda_check = false;
    if (!tried_cuda_check && hipDeviceSynchronize() != hipSuccess) {
        LOG_INFO("CUDA requested but not available on this device.");
        cuda_is_available = false;
        tried_cuda_check = true;
        return 1;  // cuda unavailable
    }

    for (int i = 0; i < device->hardware->n; ++i) {
        DXGI_ADAPTER_DESC1 desc;
        device->hardware->adapters[i]->GetDesc1(&desc);
        LOG_INFO("Adapter %d: %S", i, desc.Description);
        int res1;
        hipError_t res2;
        res2 = cudaD3D11GetDevice(&res1, device->hardware->adapters[i]);
        if (res2 != hipSuccess) {
            LOG_INFO("%d NO CUDA SUPPORT: %s | %s", i, hipGetErrorName(res2),
                     hipGetErrorString(res2));
        } else {
            LOG_INFO("%d has cuda device %d", res1);
        }
    }

    unsigned int num = 0;

    int* list = NULL;
    cudaD3D11GetDevices(&num, list, 0, device->D3D11device,
                        cudaD3D11DeviceListAll);
    LOG_INFO("num %d", num);

    hipError_t res = cudaGraphicsD3D11RegisterResource(
        &resource, device->screenshot.staging_texture, 0);
    if (res != hipSuccess) {
        LOG_ERROR("Error: %s | %s", hipGetErrorName(res),
                  hipGetErrorString(res));
    }

    return 0;
}

void dxgi_cuda_close_transfer_context() {
    if (cuda_is_available && resource) {
        hipGraphicsUnregisterResource(resource);
    }
}

int dxgi_cuda_transfer_data(CaptureDevice* device, video_encoder_t* encoder) {
    if (cuda_is_available) {
        hipError_t res;
        hipArray_t mappedArray;

        LOG_INFO("a");
        LOG_INFO("resource @ %p", resource);

        if (!resource) {
            return -1;  // failure
        }

        cudaGraphicsResourceSetMapFlags(resource, cudaGraphicsMapFlagsReadOnly);
        res = hipGraphicsMapResources(1, &resource, 0);
        if (res != hipSuccess) {
            LOG_ERROR("Error: %s | %s", hipGetErrorName(res),
                      hipGetErrorString(res));
        }

        res =
            hipGraphicsSubResourceGetMappedArray(&mappedArray, resource, 0, 0);
        if (res != hipSuccess) {
            LOG_ERROR("Error: %s | %s", hipGetErrorName(res),
                      hipGetErrorString(res));
        }

        LOG_INFO("a");

        hipGraphicsUnmapResources(1, &resource, 0);

        LOG_INFO("a");

        LOG_INFO("hwframe @ %p", encoder->hw_frame);
        LOG_INFO("data @ %p, val %p", encoder->hw_frame->data,
                 encoder->hw_frame->data[0]);
        LOG_INFO("linesize @ %p, val %d", encoder->hw_frame->linesize,
                 encoder->hw_frame->linesize[0]);
        LOG_INFO("w %d h %d", encoder->hw_frame->width,
                 encoder->hw_frame->height);
        LOG_INFO("mappedArray @ %p", mappedArray);

        hipMemcpy2DFromArray(encoder->hw_frame->data[0],
                              encoder->hw_frame->linesize[0], mappedArray, 0, 0,
                              encoder->hw_frame->width * 4,
                              encoder->hw_frame->height, hipMemcpyDefault);

        LOG_INFO("a");

        return 0;
    } else {
        return 1;  // cuda unavailable
    }
}
}